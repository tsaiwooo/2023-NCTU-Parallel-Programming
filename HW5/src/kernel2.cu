#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


__global__ void mandelKernel(float stepX, float stepY, float lowerX, float lowerY, int* ker_mem, int resX, int resY, int maxIterations,int pitch) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;



    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int globalIdy = blockIdx.y * blockDim.y + threadIdx.y;




    float x = lowerX + globalIdx * stepX;
    float y = lowerY + globalIdy * stepY;

    int rowStart = globalIdy*pitch / sizeof(int);
    int index = rowStart + globalIdx;
        
    float z_re = x, z_im = y;
    int k;
    for (k = 0; k < maxIterations; ++k)
    {
        if (z_re * z_re + z_im * z_im > 4.f)
        break;
        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = x + new_re;
        z_im = y + new_im;
    }
    ker_mem[index] = k;

}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    /*   kernel memory    */
    int *ker_mem;
    int *host_mem;
    size_t pitch;
    hipMallocPitch((void **)&ker_mem,&pitch,sizeof(int) * resX ,resY);
    hipHostAlloc((void **)&host_mem,sizeof(int) * resX * resY,hipHostMallocDefault);

    /*    Blcok setting    */
    dim3 dimGrid(resX/16,resY/16);
    dim3 dimBlock(16,16);
    
    mandelKernel<<<dimGrid,dimBlock>>>(stepX,stepY,lowerX,lowerY, ker_mem, resX, resY, maxIterations , pitch);
    hipDeviceSynchronize();

    hipMemcpy2D(host_mem,resX * sizeof(int) , ker_mem , pitch , resX * sizeof(int),resY,hipMemcpyDeviceToHost);
    // cudaMemcpy2D(host_mem, pitch , ker_mem , pitch , resX * sizeof(int) , resY, cudaMemcpyDeviceToHost);
    memcpy(img,host_mem,sizeof(int)*resX*resY);

    hipFree(ker_mem);
    hipHostFree(host_mem);
}

